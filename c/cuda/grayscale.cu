#include "hip/hip_runtime.h"
#include <stdio.h>
#include "reference_calc.cpp"
#include "utils.h"

//Introuction to Parallel Programming (Problem Set 1)
//goal: convert image from color to black and white

// How do we store pixel colors?
// RGB channels: 0 - color absent, 255 - color is completely present
// each pixel is stored as struct unsigned char 4 with
// unsigned char x, y, z, w property. w=alpha (transparency)

// How to represent color in black and white?
// (1) Intensity = (R + G + B) / 3
// Taking into account sensitivities of our eyes
// (2) Intensity = 0.299f * R + 0.589f * G + 0.144f * B

__global__ void rgba_to_grayscale(cost uchar* const rgbaImage,
                                  unsigned char* const grayImage,
                                 int numRows, int numCols){
    
}

void your_rgba_to_grayscale(const uchar* const h_rgbaImage, uchar4 * const d_rgba_image,
                           unsigned char* const d_grayImage, size_t numRows, size_t numCols) {
    const dim3 blocksize(1, 1, 1);
    const dim3 gridsize(1, 1, 1);
    
    rgba_to_grayscale<<<gridsize, blocksize>>>(d_rgba_image, d_grayImage, numRows, numCols);
    
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
}